#include <stdint.h>
#include <hip/hip_runtime.h>
__global__
void add(float *a, float *b, float *c, float n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        int idx = i * n + j;
        c[idx] = a[idx] + b[idx];
    }
}
